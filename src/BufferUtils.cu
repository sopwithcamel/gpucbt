#include <assert.h>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Buffer.h"
#include "Message.h"

namespace gpucbt {

    void Buffer::GPUSort(uint32_t num) {
        // initialize host vector
        assert(hashes_[num - 1].hash());
        assert(sizeof(messages_[num - 1]));
        thrust::device_vector<MessageHash> d_hash(hashes_, hashes_ + num);
        thrust::device_vector<Message> d_msg(messages_, messages_ + num);

        try {
            thrust::sort_by_key(d_hash.begin(), d_hash.end(), d_msg.begin());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        } catch (thrust::system_error &e) {
            fprintf(stderr, "Some other error: %s\n", e.what());
            exit(-1);
        }

        typedef thrust::device_vector<MessageHash>::iterator it_h;
        typedef thrust::device_vector<Message>::iterator it_m;
        thrust::pair<it_h, it_m> new_end;

        try {
            new_end = thrust::reduce_by_key(d_hash.begin(), d_hash.end(),
                    d_msg.begin(), d_hash.begin(), d_msg.begin(),
                    MessageHash::MessageHashComp(),
                    Message::MessageMerge());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        } catch (thrust::system_error &e) {
            fprintf(stderr, "Some other error: %s\n", e.what());
            exit(-1);
        }

        thrust::copy(d_hash.begin(), new_end.first, hashes_);
        thrust::copy(d_msg.begin(), new_end.second, messages_);
        set_num_elements(new_end.first - d_hash.begin());
    }

    bool Buffer::GPUAggregate() {
        return false;
    }
}
