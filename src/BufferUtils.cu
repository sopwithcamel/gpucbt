#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Buffer.h"
#include "Message.h"

namespace gpucbt {

    void Buffer::GPUSort(uint32_t num) {
        // initialize host vector
        thrust::device_vector<MessageHash> d_hash(hashes_, hashes_ + num);
        thrust::device_vector<Message> d_msg(messages_, messages_ + num);

        try {
            thrust::sort_by_key(d_hash.begin(), d_hash.end(), d_msg.begin());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        }

        thrust::copy(d_hash.begin(), d_hash.end(), hashes_);
        thrust::copy(d_msg.begin(), d_msg.end(), messages_);
    }

    bool Buffer::GPUAggregate() {
        return false;
    }
}
