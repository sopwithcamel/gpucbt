#include <assert.h>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Buffer.h"
#include "Message.h"

namespace gpucbt {

    void Buffer::GPUSort(uint32_t num) {
        // initialize host vector
        assert(hashes_[num - 1]);
        assert(sizeof(messages_[num - 1]));

        thrust::device_vector<uint32_t> d_hash(hashes_, hashes_ + num);
        thrust::device_vector<uint32_t> d_perm(num);

        thrust::sequence(d_perm.begin(), d_perm.begin() + num);

        try {
            thrust::sort_by_key(d_hash.begin(), d_hash.end(), d_perm.begin());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        } catch (thrust::system_error &e) {
            fprintf(stderr, "Some other error: %s\n", e.what());
            exit(-1);
        }
        // copy sorted hashes back
        thrust::copy(d_hash.begin(), d_hash.end(), hashes_);

        thrust::copy(d_perm.begin(), d_perm.end(), perm_);

/*
        typedef thrust::device_vector<uint32_t>::iterator it_h;
        typedef thrust::device_vector<uint32_t>::iterator it_p;
        thrust::pair<it_h, it_p> new_end;

        try {
            new_end = thrust::reduce_by_key(d_hash.begin(), d_hash.end(),
                    d_msg.begin(), d_hash.begin(), d_msg.begin(),
                    Message::MessageMerge());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        } catch (thrust::system_error &e) {
            fprintf(stderr, "Some other error: %s\n", e.what());
            exit(-1);
        }
*/
    }

    bool Buffer::GPUAggregate() {
        return false;
    }
}
