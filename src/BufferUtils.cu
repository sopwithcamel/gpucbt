#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Buffer.h"
#include "Message.h"

namespace gpucbt {

    void Buffer::GPUSort(uint32_t num) {
        // initialize host vector
        thrust::device_vector<Message> d(messages_, messages_ + num);

        try {
            thrust::sort(d.begin(), d.end());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        }

        thrust::copy(d.begin(), d.end(), messages_);
    }

    bool Buffer::GPUAggregate() {
        return false;
    }
}
