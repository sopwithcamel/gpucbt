#include <assert.h>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Buffer.h"
#include "Message.h"

namespace gpucbt {

    void Buffer::GPUSort(uint32_t num) {
        // initialize host vector
        assert(hashes_[num - 1]);
        assert(sizeof(messages_[num - 1]));

        // copy over hashes
        thrust::device_vector<uint32_t> d_hash(hashes_, hashes_ + num);
        // copy over messages
        thrust::device_vector<Message> d_msg(messages_, messages_ + num);

        try {
            thrust::sort_by_key(d_hash.begin(), d_hash.end(), d_msg.begin());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        } catch (thrust::system_error &e) {
            fprintf(stderr, "Some other error: %s\n", e.what());
            exit(-1);
        }

        typedef thrust::device_vector<uint32_t>::iterator it_h;
        typedef thrust::device_vector<Message>::iterator it_m;
        thrust::pair<it_h, it_m> new_end;

        try {
            new_end = thrust::reduce_by_key(d_hash.begin(), d_hash.end(),
                    d_msg.begin(), d_hash.begin(), d_msg.begin(),
                    thrust::equal_to<int>(),
                    Message::MessageMerge());
        } catch(std::bad_alloc &e) {
            fprintf(stderr, "Ran out of memory while sorting\n");
            exit(-1);
        } catch (thrust::system_error &e) {
            fprintf(stderr, "Some other error: %s\n", e.what());
            exit(-1);
        }
        // copy sorted hashes back
        thrust::copy(d_hash.begin(), new_end.first, hashes_);
        thrust::copy(d_msg.begin(), new_end.second, messages_);
        set_num_elements(new_end.first - d_hash.begin());
    }

    bool Buffer::GPUAggregate() {
        return false;
    }
}
